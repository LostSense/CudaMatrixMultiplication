#include "hip/hip_runtime.h"
#include "MatrixKer.h"

__global__ void CopyKernel(double* dest, const double* source, unsigned int size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < size)
		dest[x] = source[x];
}

__global__ void AddMatrixKernel(double* dest, const double* source, unsigned int size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < size)
		dest[x] += source[x];
}

__global__ void MultiplyMatrixKernel(const double* leftMat, const double* rightMat, double* out, size_t x, size_t y, size_t z)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < x && col < z)
	{
		for (int i = 0; i < y; ++i)
			out[row * x + col] += leftMat[row * x + i] * rightMat[i * z + col];
	}
}

hipError_t CopyMatrix(double* dest, const double* source, unsigned int size)
{
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
		return err;
	dim3 threadsPerBlock(256, 1);
	dim3 numOfBlocks(size / threadsPerBlock.x + 1, 1);
	CopyKernel <<<numOfBlocks, threadsPerBlock >>> (dest, source, size);

	return hipGetLastError();
}

hipError_t AddMatrix(double* dest, const double* source, unsigned int size)
{
	//0 - make math
	hipDeviceSynchronize();
	dim3 threadsPerBlock(256, 1);
	dim3 numOfBlocks(size / threadsPerBlock.x + 1, 1, 1);
	AddMatrixKernel << <numOfBlocks, threadsPerBlock>> > (dest, source, size);

	return hipGetLastError();
}

hipError_t MultiplyMatrix(const double* leftMat, const double* rightMat, double* outMat, size_t x, size_t y, size_t z)
{
	dim3 threadsPerBlock(16, 16);
	dim3 numOfBlocks(x / threadsPerBlock.x + 1, z / threadsPerBlock.y + 1, 1);
	MultiplyMatrixKernel << <numOfBlocks, threadsPerBlock >> > (leftMat, rightMat, outMat, x, y, z);
	return hipGetLastError();
}

hipError_t GetValue(double* src, double& out)
{
	hipDeviceSynchronize();
	hipMemcpy(&out, src, sizeof(double), hipMemcpyDeviceToHost);
	return hipGetLastError();
}

hipError_t SetValue(double* dest, double value)
{
	hipDeviceSynchronize();
	hipMemcpy(dest, &value, sizeof(double), hipMemcpyHostToDevice);
	return hipGetLastError();
}
