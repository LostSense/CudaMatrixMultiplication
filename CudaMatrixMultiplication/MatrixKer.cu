#include "hip/hip_runtime.h"
#include "MatrixKer.h"

__global__ void CopyKernel(double* dest, const double* source, unsigned int size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < size)
		dest[x] = source[x];
}

__global__ void AddMatrixKernel(double* dest, const double* source, unsigned int size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < size)
		dest[x] += source[x];
}

hipError_t CopyMatrix(double* dest, const double* source, unsigned int size)
{
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
		return err;
	dim3 threadsPerBlock(256, 1);
	dim3 numOfBlocks(size / threadsPerBlock.x + 1, 1);
	CopyKernel <<<numOfBlocks, threadsPerBlock >>> (dest, source, size);

	return hipGetLastError();
}

hipError_t AddMatrix(double* dest, const double* source, unsigned int size)
{
	//0 - make math
	hipDeviceSynchronize();
	dim3 threadsPerBlock(256, 1);
	dim3 numOfBlocks(size / threadsPerBlock.x + 1, 1, 1);
	AddMatrixKernel << <numOfBlocks, threadsPerBlock>> > (dest, source, size);

	return hipGetLastError();
}

hipError_t GetValue(double* src, double& out)
{
	hipDeviceSynchronize();
	hipMemcpy(&out, src, sizeof(double), hipMemcpyDeviceToHost);
	return hipGetLastError();
}

hipError_t SetValue(double* dest, double value)
{
	hipDeviceSynchronize();
	hipMemcpy(dest, &value, sizeof(double), hipMemcpyHostToDevice);
	return hipGetLastError();
}
