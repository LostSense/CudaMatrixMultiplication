#include "hip/hip_runtime.h"
#include "MatrixKer.h"

__global__ void CopyKernel(double* dest, double* source, size_t size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < size)
		dest[x] = source[x];
}

__global__ void AddMatrixKernel(double* dest, const double* source, size_t size)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < size)
		dest[x] += source[x];
}

hipError_t CopyMatrix(double* dest, double* source, size_t size)
{
	dim3 threadsPerBlock(256, 1);
	dim3 numOfBlocks(size / threadsPerBlock.x + 1, 1);
	CopyKernel <<<numOfBlocks, threadsPerBlock >>> (dest, source, size);

	return hipGetLastError();
}

hipError_t AddMatrix(double* dest, const double* source, size_t size)
{
	//0 - make math
	dim3 threadsPerBlock(256, 1);
	dim3 numOfBlocks(size / threadsPerBlock.x + 1, 1, 1);
	AddMatrixKernel << <numOfBlocks, threadsPerBlock>> > (dest, source, size);

	return hipGetLastError();
}

hipError_t GetValue(double* src, double& out)
{
	hipMemcpy(&out, src, sizeof(double), hipMemcpyDeviceToHost);
	return hipGetLastError();
}

hipError_t SetValue(double* dest, double value)
{
	hipMemcpy(dest, &value, sizeof(double), hipMemcpyHostToDevice);
	return hipGetLastError();
}
