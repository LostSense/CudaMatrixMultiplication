#include "hip/hip_runtime.h"
﻿
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>

#include "kernel.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] += a[i] + b[i];
}

__global__ void TwoDAddKernel(int** c, int** a, int** b)
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    c[x][y] = a[x][y] + b[x][y];
}

void TwoDAddWithCuda(int **matt, int** matt_a, int** matt_b, int size)
{
    int** dev_c = nullptr;
    int** dev_a = nullptr;
    int** dev_b = nullptr;

    hipError_t err = hipSetDevice(0);

    err = hipMalloc((void**)&dev_c, size * sizeof(int*));
    err = hipMalloc((void**)&dev_a, size * sizeof(int*));
    err = hipMalloc((void**)&dev_b, size * sizeof(int*));

    int** temp = new int*[size];
    int** temp2 = new int*[size];
    int** temp3 = new int*[size];

    for (int i = 0; i < size; ++i)
    {
        err = hipMalloc((void**)&(temp[i]), size * sizeof(int));
        err = hipMemcpy(temp[i], matt_a[i], size * sizeof(int), hipMemcpyHostToDevice);
        err = hipMalloc((void**)&(temp2[i]), size * sizeof(int));
        err = hipMemcpy(temp2[i], matt_b[i], size * sizeof(int), hipMemcpyHostToDevice);
        err = hipMalloc((void**)&(temp3[i]), size * sizeof(int));
        err = hipMemcpy(temp3[i], matt[i], size * sizeof(int), hipMemcpyHostToDevice);
    }

    err = hipMemcpy(dev_a, temp, size * sizeof(int*), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_b, temp2, size * sizeof(int*), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_c, temp3, size * sizeof(int*), hipMemcpyHostToDevice);

    dim3 tpb(size, size);
    TwoDAddKernel<<<1, tpb >>> (dev_c, dev_a, dev_b);
    err = hipGetLastError();
    err = hipDeviceSynchronize();

    for (int i = 0; i < size; ++i)
    {
        err = hipMemcpy(matt[i], temp3[i], size * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(temp[i]);
        hipFree(temp2[i]);
        hipFree(temp3[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

void TwoDAddWithCuda2(int **matt, int** matt_a, int** matt_b, int size)
{
    int** dev_c = nullptr;
    int** dev_a = nullptr;
    int** dev_b = nullptr;

    hipError_t err = hipSetDevice(0);

    err = hipMalloc((void**)&dev_c, size * sizeof(int*));
    err = hipMalloc((void**)&dev_a, size * sizeof(int*));
    err = hipMalloc((void**)&dev_b, size * sizeof(int*));

    int** temp = new int*[size];
    int** temp2 = new int*[size];
    int** temp3 = new int*[size];

    for (int i = 0; i < size; ++i)
    {
        err = hipMalloc((void**)&(temp[i]), size * sizeof(int));
        err = hipMemcpy(temp[i], matt_a[i], size * sizeof(int), hipMemcpyHostToDevice);
        err = hipMalloc((void**)&(temp2[i]), size * sizeof(int));
        err = hipMemcpy(temp2[i], matt_b[i], size * sizeof(int), hipMemcpyHostToDevice);
        err = hipMalloc((void**)&(temp3[i]), size * sizeof(int));
        err = hipMemcpy(temp3[i], matt[i], size * sizeof(int), hipMemcpyHostToDevice);
    }

    err = hipMemcpy(dev_a, temp, size * sizeof(int*), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_b, temp2, size * sizeof(int*), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_c, temp3, size * sizeof(int*), hipMemcpyHostToDevice);

    dim3 tpb(size, size);
    TwoDAddKernel<<<1, tpb >>> (dev_c, dev_a, dev_b);
    err = hipGetLastError();
    err = hipDeviceSynchronize();

    for (int i = 0; i < size; ++i)
    {
        err = hipMemcpy(matt[i], temp3[i], size * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(temp[i]);
        hipFree(temp2[i]);
        hipFree(temp3[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    hipMemset(dev_c, 0, size * sizeof(int));

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
